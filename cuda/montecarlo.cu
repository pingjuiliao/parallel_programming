#include "hip/hip_runtime.h"
// Array multiplication: C = A * B:

// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

#ifndef BLOCKSIZE
#define BLOCKSIZE		32		// number of threads per block
#endif

#ifndef SIZE
#define SIZE			1*8096	// array size
#endif

#ifndef NUMTRIALS
#define NUMTRIALS		100		// to make the timing more accurate
#endif

#ifndef TOLERANCE
#define TOLERANCE		0.00001f	// tolerance to relative error
#endif

const float XCMIN = 0.0 ;
const float XCMAX = 2.0 ;
const float YCMIN = 0.0 ;
const float YCMAX = 2.0 ;
const float RMIN  = 0.5 ;
const float RMAX  = 2.0 ;

// function prototypes 
float Ranf(float, float);

// array multiplication (CUDA Kernel) on the device: C = A * B

__global__  void ArrayMul( float *A, float *B, float *C )
{
	__shared__ float prods[BLOCKSIZE];

	unsigned int numItems = blockDim.x;
	unsigned int tnum = threadIdx.x;
	unsigned int wgNum = blockIdx.x;
	unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;

	prods[tnum] = A[gid] * B[gid];

	for (int offset = 1; offset < numItems; offset *= 2)
	{
		int mask = 2 * offset - 1;
		__syncthreads();
		if ((tnum & mask) == 0)
		{
			prods[tnum] += prods[tnum + offset];
		}
	}

	__syncthreads();
	if (tnum == 0)
		C[wgNum] = prods[0];
}


__global__ void MonteCarlo( float *xcs, float *ycs, float *rs, unsigned int *p_numHits )  
{	
	__shared__ unsigned int hits[BLOCKSIZE];
	unsigned int numItems = blockDim.x;
	unsigned int tnum = threadIdx.x;
	unsigned int wgNum = blockIdx.x;
	unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;
	
	// c numHits == 0

	// randomize the location and radius of the circle:
	float xc = xcs[gid];
	float yc = ycs[gid];
	float  r =  rs[gid];

	// solve for the intersection using the quadratic formula:
	float a = 2.;
	float b = -2.*( xc + yc );
	float c = xc*xc + yc*yc - r*r;
	float d = b*b - 4.*a*c;
	// If d is less than 0., then the circle was completely missed. (Case A) Continue on to the next trial in the for-loop.
	if ( d >= 0. ) {

		// hits the circle:
		// get the first intersection:
		d = sqrt( d );
		float t1 = (-b + d ) / ( 2.*a );	// time to intersect the circle
		float t2 = (-b - d ) / ( 2.*a );	// time to intersect the circle
		float tmin = t1 < t2 ? t1 : t2;		// only care about the first intersection
		// If tmin is less than 0., then the circle completely engulfs the laser pointer. (Case B) Continue on to the next trial in the for-loop.
		if ( tmin >= 0. ) {


			// where does it intersect the circle?
			float xcir = tmin;
			float ycir = tmin;

			// get the unitized normal vector at the point of intersection:
			float nx = xcir - xc;
			float ny = ycir - yc;
			float n = sqrt( nx*nx + ny*ny );
			nx /= n;	// unit vector
			ny /= n;	// unit vector

			// get the unitized incoming vector:
			float inx = xcir - 0.;
			float iny = ycir - 0.;
			float in = sqrt( inx*inx + iny*iny );
			inx /= in;	// unit vector
			iny /= in;	// unit vector

			// get the outgoing (bounced) vector:
			float dot = inx*nx + iny*ny;
			// float outx = inx - 2.*nx*dot;	// angle of reflection = angle of incidence`
			float outy = iny - 2.*ny*dot;	// angle of reflection = angle of incidence`

			// find out if it hits the infinite plate:
			float t = ( 0. - ycir ) / outy;
			// If t is less than 0., then the reflected beam went up instead of down. Continue on to the next trial in the for-loop.
			if ( t >= 0. ) {
				hits[tnum] ++;
			}
			//  Otherwise, this beam hit the infinite plate. (Case D) Increment the number of hits and continue on to the next trial in the for-loop.

		}
	}
	for (int offset = 1; offset < numItems; offset *= 2)
	{
		int mask = 2 * offset - 1;
		__syncthreads();
		if ((tnum & mask) == 0)
		{
			hits[tnum] += hits[tnum + offset];
		}
	}

	__syncthreads();
	if (tnum == 0)
		p_numHits[wgNum] = hits[0];
}

// main program:

	int
main( int argc, char* argv[ ] )
{
	int dev = findCudaDevice(argc, (const char **)argv);

	// allocate host memory:

	float * hA = new float [ SIZE ];
	float * hB = new float [ SIZE ];
	float * hC = new float [ SIZE ];
	unsigned int * hR = new unsigned int [ SIZE/BLOCKSIZE ]; // R for results
	
	for( int i = 0; i < SIZE; i++ )
	{
		hA[i] = Ranf( XCMIN, XCMAX ) ;
		hB[i] = Ranf( YCMIN, YCMAX ) ;
		hC[i] = Ranf( RMIN, RMAX ) ;	
	}
	for ( int i = 0; i < SIZE/BLOCKSIZE; ++i ) {
		hR[i] = 0;
	}

	// allocate device memory:

	float *dA, *dB, *dC;
	unsigned int *dR ;

	dim3 dimsA( SIZE, 1, 1 );
	dim3 dimsB( SIZE, 1, 1 );
	dim3 dimsC( SIZE, 1, 1 );
	dim3 dimsR( SIZE/BLOCKSIZE, 1, 1 );
	//__shared__ float prods[SIZE/BLOCKSIZE];


	hipError_t status;
	status = hipMalloc( reinterpret_cast<void **>(&dA), SIZE*sizeof(float) );
	checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dB), SIZE*sizeof(float) );
	checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dC), (SIZE)*sizeof(float) );
	checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dR), (SIZE/BLOCKSIZE)*sizeof(unsigned int));
	checkCudaErrors( status );
	// copy host memory to the device:

	status = hipMemcpy( dA, hA, SIZE*sizeof(float), hipMemcpyHostToDevice );
	checkCudaErrors( status );
	status = hipMemcpy( dB, hB, SIZE*sizeof(float), hipMemcpyHostToDevice );
	checkCudaErrors( status );
	status = hipMemcpy( dC, hC, SIZE*sizeof(float), hipMemcpyHostToDevice );
	checkCudaErrors( status );
	// setup the execution parameters:

	dim3 threads(BLOCKSIZE, 1, 1 );
	dim3 grid( SIZE / threads.x, 1, 1 );

	// Create and start timer

	hipDeviceSynchronize( );

	// allocate CUDA events that we'll use for timing:

	hipEvent_t start, stop;
	status = hipEventCreate( &start );
	checkCudaErrors( status );
	status = hipEventCreate( &stop );
	checkCudaErrors( status );

	// record the start event:

	status = hipEventRecord( start, NULL );
	checkCudaErrors( status );

	// execute the kernel:
	for( int t = 0; t < NUMTRIALS; t++)
	{
		MonteCarlo<<< grid, threads >>>( dA, dB, dC, dR );
	}
	// record the stop event:

	status = hipEventRecord( stop, NULL );
	checkCudaErrors( status );

	// wait for the stop event to complete:

	status = hipEventSynchronize( stop );
	checkCudaErrors( status );

	float msecTotal = 0.0f;
	status = hipEventElapsedTime( &msecTotal, start, stop );
	checkCudaErrors( status );

	// compute and print the performance

	double secondsTotal = 0.001 * (double)msecTotal;
	double multsPerSecond = (float)SIZE * (float)NUMTRIALS / secondsTotal;
	double megaMultsPerSecond = multsPerSecond / 1000000.;
	fprintf( stderr, "Array Size = %10d, MegaMultReductions/Second = %10.2lf\n", SIZE, megaMultsPerSecond );

	// copy result from the device to the host:

	status = hipMemcpy( hR, dR, (SIZE/BLOCKSIZE)*sizeof(unsigned int), hipMemcpyDeviceToHost );
	checkCudaErrors( status );

	// check the sum :

	unsigned int sum = 0;
	for(int i = 0; i < SIZE/BLOCKSIZE; i++ )
	{
		//fprintf(stderr, "hC[%6d] = %10.2f\n", i, hC[i]);
		sum += hR[i];
	}
	fprintf( stderr, "\nsum = %u\n", sum );

	// clean up memory:
	delete [ ] hA;
	delete [ ] hB;
	delete [ ] hC;
	delete [ ] hR;

	status = hipFree( dA );
	checkCudaErrors( status );
	status = hipFree( dB );
	checkCudaErrors( status );
	status = hipFree( dC );
	checkCudaErrors( status );
	status = hipFree( dR );
	checkCudaErrors( status );

	return 0;
}

float
Ranf( float low, float high ) {
	float r = (float) rand() ;
	float t = r / (float) RAND_MAX ;
	return low + t * ( high - low ) ;
}
